#include "hip/hip_runtime.h"
/*
 * The Smith-Waterman algorithm, is a dynamic programming algorithm were the DP matrices 
 * involved in the computation are calculated dynamically. There are 3 DP: M, X, and Y 
 * each contributing a score from one of the three directions an entry in the SW scoring 
 * matrix can obtain. With the SW algorithm we implement affine-gap penalty scoring, thus
 * working towards a local alignment algortihm with affine-gap penalty as in the seed 
 * extension stage of BWA-MEM sequencing algorithm. 
 */
 #include "stdio.h"
 #include "string.h"
 #include "stdlib.h"
 #include "stdint.h"
 #include "swalign.h"
 #include <iostream>
 #include "hip/hip_runtime.h"

/*const int for penalty*/
const int penalty = gap_open + gap_extn;    

__host__ __device__ void init_DP(int M[][L+1], int X[][L+1], int Y[][L+1]){
	M[0][0] = 0;
	X[0][0] = -1000;
	Y[0][0] = -1000;
	for(int i=1; i <L+1; i++){
		M[i][0] = 0;
		X[i][0] = -1000;   //Just a large negative number
		Y[i][0] = -1000;
	}

	for(int j=1; j< L+1; j++){
		M[0][j] = 0;
		X[0][j] = -1000;   //Just a large negative number
		Y[0][j] = -1000;
	}
}

__global__ void read_align(char *seq1, char *seq2, char *seq1_out, char *seq2_out){
    
   int seq_i;
   sw_entry Score_Matrix[L+1][L+1];
   int M[L+1][L+1], X[L+1][L+1], Y[L+1][L+1];  //DP matrices
   int A, B, S_I;

   int index = blockIdx.x * blockDim.x +threadIdx.x;
   
   if(index < no_seq)
   {   
        seq_i = index * (L+1);
        seq1[seq_i] = '-';
        seq2[seq_i] = '-';
        seq1_out[seq_i] = '$';
        seq2_out[seq_i] = '$';
        /*Start scoring*/
       
        init_DP(M, X, Y);
        
            Score_Matrix[0][0].value = 0;
            for(int j=1; j<L+1; j++){
              Score_Matrix[0][j].value = 0;
            }
            for(int i=1; i<L+1; i++){
              Score_Matrix[i][0].value = 0;
            }
        //A = M[0][0];
       //seq1_out[A] = 'Z';
	   
 /*Compute DP matrices */
    int M_max =0, X_max, Y_max;
    int M_x, M_y, M_m;
    int match_score;
    int si, sj;

    for(int I = 1; I < L+1; I++){
       for(int J = 1; J <L+1; J++){
			  si = I + seq_i;
			  sj = J + seq_i;
	   if(seq1[si] == seq2[sj])
		match_score = match;
	   else
		match_score = mismatch;
           
	   M_m = M[I-1][J-1] + match_score;
	   M_x = X[I-1][J-1] + match_score;
	   M_y = Y[I-1][J-1] + match_score;

		if(M_m >= M_x && M_m >= M_y && M_m > 0) 
			M_max = M_m;
		else if(M_x >= M_m && M_x >= M_y && M_x > 0)
			M_max = M_x;
		     else if(M_y >= M_m && M_y >= M_x && M_y > 0)
			     M_max = M_y;

		M[I][J] =  M_max;
         
	    Y_max = gap_extn + Y[I][J-1];
	    if(penalty + M[I][J-1] > Y_max)
		Y_max = M[I][J-1] + penalty;

	    Y[I][J] = Y_max;

	    X_max = gap_extn + X[I-1][J];
	    if(penalty + M[I-1][J] > X_max)
		X_max = M[I-1][J] + penalty;

	    X[I][J] = X_max;


	    if(X_max >= Y_max && X_max >= M_max){
		Score_Matrix[I][J].value = X_max;
		Score_Matrix[I][J].direction = x;
	    }
	    else if(Y_max >= X_max && Y_max >= M_max){
		    Score_Matrix[I][J].value = Y_max;
		    Score_Matrix[I][J].direction = y;
		 }
		 else if(M_max >= X_max && M_max >= Y_max){
			 Score_Matrix[I][J].value = M_max;
			 Score_Matrix[I][J].direction = m;
		 }
     
         
	} 
      }
                
        //A = Score_Matrix[0][0].value;
	//seq1_out[A] = 'Y';
/*Maximum Score in SW matrix*/
  
	sw_entry sw_max;
	int val;

	sw_max = Score_Matrix[0][0];
	for(int i=0; i < L+1; i++){
		for(int j=0; j < L+1; j++){
			val = Score_Matrix[i][j].value;
			if(val > sw_max.value){
				sw_max.value = val;
				A = i;
				B = j;
				if(i >= j)
				  S_I = i;
				else
				  S_I = j;
			}
		}
          }
	//A = Score_Matrix[0][0].value;
        //seq2_out[B] = 'W';
	
   /*Traceback function*/
    
     DP_dir SW_dir;
     char c1, c2; 
     
     for(int n = L; n >=0; --n){
	if(M[A][B]!=0 && n <= S_I){  
       		SW_dir = Score_Matrix[A][B].direction;   
    		if(SW_dir == m){
                	c1 = seq1[A + seq_i];
    			c2 = seq2[B + seq_i];
    			A = A-1;
    			B = B-1;
    		} else if(SW_dir == x){
    		        c2 = '-'; 
    		   	c1 = seq1[A + seq_i];
    		   	A = A-1;
    			}
    	       		else if(SW_dir == y){
    	       	      		c1 = '-';
    	       	      		c2 = seq2[B + seq_i];
    	       	      		B = B-1;
    	            		}
		seq1_out[n + seq_i] = c1;
	        seq2_out[n + seq_i] = c2;
       } 
	 else if(M[A][B] == 0  && n <=S_I){//((M[A][B] != 0 && n > S_I)  || (M[A][B] == 0 && n <= S_I)){
		seq1_out[n + seq_i] = '.';
	        seq2_out[n + seq_i] = '.';
	     }else if(M[A][B] !=0 && n >S_I){
		seq1_out[n + seq_i] = '*';
	        seq2_out[n + seq_i] = '*';
	     }	
     
     }
             

    }
}


/*Main function*/
int main(int argc, char *argv[]){
    
    FILE *input1, *input2;
    FILE *output;
   /*Read in the two sequences to be aligned, one from refrence and another a query
    *short read, which are stored in a text file and store in arrays seq1[] and seq2[]
    */
    //sprintf(buff1,argv[1]);
    //sprintf(buff2,argv[2]);
    input1 = fopen("seq1_out.txt","rb");//input1 = fopen(argv[1],"rb");
	if (!input1) {
	  printf("Unable to open input file %s.\n", "seq1_out.txt");//argv[1]);
	  fflush(stdout);
	  exit(-1);
	}	
	input2 = fopen("seq2_out.txt","rb");//input2 = fopen(argv[2],"rb");
	if (!input2) {
	  printf("Unable to open input file %s.\n", "seq2_out.txt");//argv[2]);
	  fflush(stdout);
	  exit(-1);
	}

    output = fopen("align_out.txt","wb");
    
    char *seq1, *seq2;
    char *seq1_out, *seq2_out;
    char line[] = "Output seq 1:";
    char line1[] = "Output seq 2:";
    char head[] = "Sequence pair";
    int l_size = strlen(line);
    size_t  s_size = no_seq * (L+1) * sizeof(char) ;
   

    /*Dynamic memory allocation at Host*/
    seq1 = (char*) malloc(s_size);
    if (seq1 == NULL) fprintf(stderr, "Bad malloc on seq1\n");
    seq2 = (char*) malloc(s_size);
    if (seq2 == NULL) fprintf(stderr, "Bad malloc on seq2\n");
    seq1_out = (char*) malloc(s_size);
    if (seq1_out == NULL) fprintf(stderr, "Bad malloc on seq1_out\n");
    seq2_out = (char*) malloc(s_size);
    if (seq2_out == NULL) fprintf(stderr, "Bad malloc on seq2_out\n");
   
    /*Allocate memory in Device*/
    char *seq1_d;
    hipMalloc(&seq1_d, s_size);
    char *seq2_d;
    hipMalloc(&seq2_d, s_size);
    char *seq1_out_d;
    hipMalloc(&seq1_out_d, s_size);
    char *seq2_out_d;
    hipMalloc(&seq2_out_d, s_size);

    /* Load data from textfile */
    seq1[0] = '-';
    seq2[0] = '-';
    fread(&seq1[1], sizeof(char), ((L+1)*(no_seq-1)+ L), input1);
    fread(&seq2[1], sizeof(char), ((L+1)*(no_seq-1)+ L), input2);
     
   // printf("First char of seq1:%c, seq2:%c, last char of seq1:%c, seq2:%c\n", seq1[1], seq2[1], seq1[L], seq2[L]);

    fclose(input1);
    fclose(input2);
    fflush(stdout);
    
    //printf("Strlen of seq1:%d, seq2:%d\n", strlen(seq1), strlen(seq2));
    /*Copy data from Host to Device*/
    hipMemcpy(seq1_d, seq1, s_size, hipMemcpyHostToDevice);
    hipMemcpy(seq2_d, seq2, s_size, hipMemcpyHostToDevice);
   
    /*Perform alignment at Device*/
    read_align<<<1,no_seq>>>(seq1_d, seq2_d, seq1_out_d, seq2_out_d);
  
    hipDeviceSynchronize();
   
    /*Copy output data from Device to Host*/
    hipMemcpy(seq1_out, seq1_out_d, s_size, hipMemcpyDeviceToHost);
    hipMemcpy(seq2_out, seq2_out_d, s_size, hipMemcpyDeviceToHost);
    hipMemcpy(seq1, seq1_d, s_size, hipMemcpyDeviceToHost);
    hipMemcpy(seq2, seq2_d, s_size, hipMemcpyDeviceToHost);
    //printf("Strlen of seq1_out:%d, seq2_out:%d\n",strlen(seq1_out), strlen(seq2_out));
    /* Write result to file */
    for(int m=0; m < no_seq; m++){
	fwrite(head, sizeof(char), strlen(head), output);
        fprintf(output, "%d\n", m);	
        fwrite(line, sizeof(char), strlen(line), output);
        //fwrite(&seq1[m*(L+1)], sizeof(char), L+1, output);
        //fprintf(output,"\n");
        fwrite(&seq1_out[m*(L+1)], sizeof(char), L+1, output);
        fprintf(output,"\n");
        fwrite(line1, sizeof(char), strlen(line1), output);
        //fwrite(&seq2[m*(L+1)], sizeof(char), L+1, output);
        //fprintf(output, "\n");
        fwrite(&seq2_out[m*(L+1)], sizeof(char), L+1, output);
        if(m != no_seq-1)
          fprintf(output,"\n");
    }

	fclose(output);

	printf("Output complete.\n");
	fflush(stdout);

    /*Free Device memory*/
    hipFree(seq1_d);
    hipFree(seq2_d);
    hipFree(seq1_out_d);
    hipFree(seq2_out_d);

    /*Free Host memory*/
    free(seq1);
    free(seq2);
    free(seq1_out);
    free(seq2_out);
}
